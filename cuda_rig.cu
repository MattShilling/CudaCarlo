#include "cuda_rig.h"

#include <algorithm>
#include <cmath>
#include <omp.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

void CudaRig::Init() {
  // Initialize test memory.
  test_init_(mem_);
}

int CudaRig::InitAndCopy(void **device, void *host, size_t sz) {
  hipError_t status;
  status = hipMalloc(device, sz);
  checkCudaErrors(status);
  // Copy host memory to the GPU.
  status =
      hipMemcpy(*device, host, sz, hipMemcpyHostToDevice);
  checkCudaErrors(status);

  return status;
}

void CudaRig::StartCudaTimer(CudaTimer *t) {
  hipError_t status;
  // Create and start timer.
  hipDeviceSynchronize();

  // Allocate CUDA events that we'll use for timing.
  status = hipEventCreate(&(t->start));
  checkCudaErrors(status);
  status = hipEventCreate(&(t->stop));
  checkCudaErrors(status);

  // Record the start event.
  status = hipEventRecord(t->start, NULL);
  checkCudaErrors(status);
}

void CudaRig::StopCudaTimer(CudaTimer *t){
  hipError_t status;
  // Record the stop event.
  status = hipEventRecord(t->stop, NULL);
  checkCudaErrors(status);

  // Wait for the stop event to complete.
  status = hipEventSynchronize(t->stop);
  checkCudaErrors(status);
}